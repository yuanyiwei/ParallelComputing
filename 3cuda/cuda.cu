#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
//CUDA RunTime API
#include <hip/hip_runtime.h> //单个block大小
#define THREAD_NUM 256    ///矩阵大小
#define MATRIX_SIZE 1000  ///block个数
int blocks_num = (MATRIX_SIZE + THREAD_NUM - 1) / THREAD_NUM;
int main()
{ //定义矩阵
    float *a, *b, *c, *d;
    int n = MATRIX_SIZE; //分配主机端内存
    a = (float *)malloc(sizeof(float) * n * n);
    b = (float *)malloc(sizeof(float) * n * n);
    c = (float *)malloc(sizeof(float) * n * n);
    d = (float *)malloc(sizeof(float) * n * n);
    float *cuda_a, *cuda_b, *cuda_c;
    //分配设备端显存
    hipMalloc((void **)&cuda_a, sizeof(float) * n * n);
    hipMalloc((void **)&cuda_b, sizeof(float) * n * n);
    hipMalloc((void **)&cuda_c, sizeof(float) * n * n);
    ///生成矩阵a, b
    generateMatrix(a, b);
    //hipMemcpyHostToDevice - 从内存复制到显存
    //hipMemcpyDeviceToHost - 从显存复制到内存
    hipMemcpy(cuda_a, a, sizeof(float) * n * n, hipMemcpyHostToDevice);
    hipMemcpy(cuda_b, b, sizeof(float) * n * n, hipMemcpyHostToDevice);
    ///设备端函数
    CUDAkernal<<<blocks_num, THREAD_NUM, 0>>>(cuda_a, cuda_b, cuda_c, n, time);
    //hipMemcpy 将结果从显存中复制回内存
    hipMemcpy(c, cuda_c, sizeof(float) * n * n, hipMemcpyDeviceToHost);
    //Free
    hipFree(cuda_a);
    hipFree(cuda_b);
    hipFree(cuda_c);
}

__global__ static void CUDAkernal(const float *a, const float *b, float *c, int n)
{
    //block内的threadID
    const int tid = threadIdx.x;
    //blockID
    const int bid = blockIdx.x;
    //全局threadID
    const int idx = bid * THREAD_NUM + tid;
    const int row = idx / n;
    const int column = idx % n;
    //计算矩阵乘法
    if (row < n && column < n)
    {
        float t = 0;
        for (i = 0; i < n; i++)
        {
            t += a[row * n + i] * b[i * n + column];
        }
        c[row * n + column] = t;
    }
}