
#include <hip/hip_runtime.h>
#include <cstdio>
#include <fstream>
#include <iomanip>
#include <iostream>

hipEvent_t start, stop;
float elapsedTime = 0.0;
constexpr auto N = 1000;
constexpr auto BlkNum = 100;

__global__ void dot(double *a, double *b, double *res);

int main()
{
  std::ios::sync_with_stdio(false);
  std::ifstream in("in.txt");
  if (!in)
  {
    std::cerr << "Getting input failed\n";
    return -2;
  }
  auto a = new double[N * N], b = new double[N * N], res = new double[N * N];
  for (auto i = 0; i < N; i++)
  {
    for (auto j = 0; j < N; j++)
    {
      in >> a[i * N + j];
    }
  }
  for (auto i = 0; i < N; i++)
  {
    for (auto j = 0; j < N; j++)
    {
      in >> b[i * N + j];
    }
  }
  in.close();

  double *a1, *b1, *res1;
  hipMalloc(&a1, sizeof(double) * N * N);
  hipMemcpy(a1, a, sizeof(double) * N * N, hipMemcpyHostToDevice);
  hipMalloc(&b1, sizeof(double) * N * N);
  hipMemcpy(b1, b, sizeof(double) * N * N, hipMemcpyHostToDevice);
  hipMalloc(&res1, sizeof(double) * N * N);

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  dot<<<BlkNum, N / BlkNum>>>(a1, b1, res1);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  hipMemcpy(res, res1, sizeof(double) * N * N, hipMemcpyDeviceToHost);
  std::cout << "Running Time: " << elapsedTime << "s" << std::endl;

  std::ofstream out("out.txt");
  if (!out)
  {
    std::cerr << "Getting output failed\n";
    return -1;
  }
  for (auto i = 0; i < N; i++)
  {
    for (auto j = 0; j < N; j++)
    {
      out << std::setprecision(15) << res[i * N + j] << ",";
    }
    out << "\n";
  }
  out.close();
  free(a);
  free(b);
  free(res);
  hipFree(a1);
  hipFree(b1);
  hipFree(res1);
  return 0;
}

__global__ void dot(double *a, double *b, double *res)
{
  auto i = blockIdx.x * blockDim.x + threadIdx.x;
  for (auto j = 0; j < N; j++)
  {
    res[i * N + j] = 0;
    for (auto k = 0; k < N; k++)
    {
      res[i * N + j] += a[i * N + k] * b[k * N + j];
    }
  }
}
