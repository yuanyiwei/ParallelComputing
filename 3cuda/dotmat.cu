
#include <hip/hip_runtime.h>
#include <cstdio>
#include <fstream>
#include <iomanip>
#include <iostream>

hipEvent_t start, stop;
float elapsedTime = 0.0;
constexpr auto N = 1000;
constexpr auto BlkNum = 100;

__global__ void dot(double *a, double *b, double *c)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    for (int j = 0; j < N; j++)
    {
        c[i * N + j] = 0;
        for (int k = 0; k < N; k++)
            c[i * N + j] += a[i * N + k] * b[k * N + j];
    }
}

int main()
{
    std::ios::sync_with_stdio(false);
    std::ifstream in("in.txt");
    if (!in)
    {
        std::cerr << "Err: input\n";
        return -2;
    }
    auto a = new double[N * N], b = new double[N * N], res = new double[N * N];
    for (int i = 0; i < N; i++)
        for (int j = 0; j < N; j++)
            in >> a[i * N + j];
    for (int i = 0; i < N; i++)
        for (int j = 0; j < N; j++)
            in >> b[i * N + j];
    in.close();

    double *a1, *b1, *res1;
    hipMalloc(&a1, sizeof(double) * N * N);
    hipMemcpy(a1, a, sizeof(double) * N * N, hipMemcpyHostToDevice);
    hipMalloc(&b1, sizeof(double) * N * N);
    hipMemcpy(b1, b, sizeof(double) * N * N, hipMemcpyHostToDevice);
    hipMalloc(&res1, sizeof(double) * N * N);

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    dot<<<BlkNum, N / BlkNum>>>(a1, b1, res1);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(res, res1, sizeof(double) * N * N, hipMemcpyDeviceToHost);
    std::cout << "Running Time: " << elapsedTime << "s" << std::endl;

    std::ofstream out("out.txt");
    if (!out)
    {
        std::cerr << "Err: output\n";
        return -1;
    }
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
            out << std::setprecision(15) << res[i * N + j] << ",";
        out << "\n";
    }
    out.close();
    free(a);
    free(b);
    free(res);
    hipFree(a1);
    hipFree(b1);
    hipFree(res1);
    return 0;
}
